
#include <hip/hip_runtime.h>
extern "C" __global__ void VecAdd_kernel(
    const float *A, const float *B, float *C, int N) {

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
        C[i] = A[i] + B[i];
  }
}