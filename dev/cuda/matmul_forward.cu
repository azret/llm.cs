#include "hip/hip_runtime.h"
/*
Kernels for matmul forward pass.
It's advised to use OpenMP here because the CPU implementation is fairly slow otherwise

Compile example:
nvcc -O3 --use_fast_math -Xcompiler -fopenmp matmul_forward.cu -o matmul_forward -lcublas -lcublasLt

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
OMP_NUM_THREADS=32 ./matmul_forward 1

version 2 calls cuBLAS, very fast
OMP_NUM_THREADS=32 ./matmul_forward 2

version 3 calls cuBLASLt, should be even faster
OMP_NUM_THREADS=32 ./matmul_forward 3
*/

#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <omp.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CUDA setup

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static hipblasHandle_t cublas_handle;
static hipblasLtHandle_t cublaslt_handle;
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;

// ----------------------------------------------------------------------------
// CPU code reference

void matmul_forward_cpu(float* out,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C, int OC) {
    // OC is short for "output channels"
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // out will be (B,T,OC)
    #pragma omp parallel for collapse(2)
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float* out_bt = out + b * T * OC + t * OC;
            const float* inp_bt = inp + b * T * C + t * C;
            for (int o = 0; o < OC; o++) {
                float val = (bias != NULL) ? bias[o] : 0.0f;
                const float* wrow = weight + o*C;
                for (int i = 0; i < C; i++) {
                    val += inp_bt[i] * wrow[i];
                }
                out_bt[o] = val;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// kernel 1: naive kernel, every thread handles one output element, direct global memory access
__global__ void matmul_forward_kernel1(float* out,
                                       const float* inp, const float* weight, const float* bias,
                                       int BT, int C, int OC) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // in the naive kernel, every thread handles one element of out
    int bt = blockIdx.x * blockDim.x + threadIdx.x;
    int oc = blockIdx.y * blockDim.y + threadIdx.y;
    if (bt < BT && oc < OC) {
        int b = bt / BT;
        int t = bt % BT;
        float val = (bias != NULL) ? bias[oc] : 0.0f;
        const float* wrow = weight + oc*C;
        const float* inp_bt = inp + b * BT * C + t * C;
        for (int i = 0; i < C; i++) {
            val += inp_bt[i] * wrow[i];
        }
        out[bt * OC + oc] = val;
    }
}

// is there no better way other than just adding bias with a whole separate kernel?
// this is a highly memory-bound operation, should be fused into the matmul kernel
// but i can't seem to find a cuBLAS function that does this
__global__ void add_bias(float* out, const float* bias, int B, int T, int OC) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < B * T * OC; i += stride) {
        int col = i % OC;
        out[i] += bias[col];
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

// kernel 1 is the most naive matmul kernel
void matmul_forward1(float* out,
                     const float* inp, const float* weight, const float* bias,
                     int B, int T, int C, int OC,
                     const int sqrt_block_size) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    dim3 gridDim(ceil_div(B * T, sqrt_block_size), ceil_div(OC, sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);
    matmul_forward_kernel1<<<gridDim, blockDim>>>(out, inp, weight, bias, B*T, C, OC);
    cudaCheck(hipGetLastError());
}

// kernel 2 calls cuBLAS, which should be very efficient
void matmul_forward2(float* out,
                     const float* inp, const float* weight, const float* bias,
                     int B, int T, int C, int OC,
                     const int sqrt_block_size) {
    // for reference API is:
    // hipblasStatus_t hipblasSgemm(hipblasHandle_t handle,
    //                        hipblasOperation_t transa, hipblasOperation_t transb,
    //                        int m, int n, int k,
    //                        const float           *alpha,
    //                        const float           *A, int lda,
    //                        const float           *B, int ldb,
    //                        const float           *beta,
    //                        float           *C, int ldc)
    // for us, inp is (B*T, C), weight is (OC, C), out is (B*T, OC)
    // cuBLAS does C = alpha * A * B + beta * C
    // where A is mxk, B is kxn, C is mxn
    // now, because we use row-major storage, cuBLAS (which is column-major) sees our matrices transposed.
    // algorithmically / in e.g. PyTorch we want to do: out = inp @ weight.T
    // but because cuBLAS is column-major, we actually want to get it to calculate out.T . Mathematically, this is:
    // out.T = weight @ inp.T
    // but again, our variables look transposed, so using the actual weight/inp we have here in this function, this becomes
    // out.T = weight.T @ inp
    // so we need to get cuBLAS to calculate weight.T @ inp (the variables here are the actual ones in this function)
    // => need to call cuBLAS with A = weight, B = inp
    // => need to call cuBLAS with transa = HIPBLAS_OP_T, transb = HIPBLAS_OP_N

    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B*T, C, &alpha, weight, C, inp, C, &beta, out, OC));
    // and now we still have to add the bias... (ew)
    if (bias != NULL) {
        int block_size = sqrt_block_size * sqrt_block_size;
        int grid_size = ceil_div(OC * B * T, block_size);
        add_bias<<<grid_size, block_size>>>(out, bias, B, T, OC);
        cudaCheck(hipGetLastError());
    }
}

// uses cublasLt to fuse the bias and gelu
// https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu
void matmul_forward3(float* out,
                     const float* inp, const float* weight, const float* bias,
                     int B, int T, int C, int OC) {
    int has_bias = (bias != NULL);
    int has_gelu = 0;

    // check bias alignment
    if(((uintptr_t)bias % 16) != 0) {
        printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
        exit(EXIT_FAILURE);
    }

    int returnedResults = 0;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayout;
    hipblasLtMatrixLayout_t inputLayout;
    hipblasLtMatrixLayout_t outputLayout;
    hipblasLtMatrixLayout_t biasLayout;
    hipblasLtMatmulHeuristicResult_t heuristic;

    // create the operation descriptor
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_DEFAULT;
    if (has_bias && has_gelu) {
        epilogueBias = HIPBLASLT_EPILOGUE_GELU_BIAS;
    } else if (has_bias) {
        epilogueBias = HIPBLASLT_EPILOGUE_BIAS;
    } else if (has_gelu) {
        epilogueBias = HIPBLASLT_EPILOGUE_GELU;
    }
    cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute_type, HIP_R_32F));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    // define matrix layouts
    cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, HIP_R_32F, C, OC, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, HIP_R_32F, C, B*T, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, HIP_R_32F, OC, B*T, OC));
    cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, HIP_R_32F, OC, 1, OC));

    // create a preference handle with specified max workspace
    cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
    cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    // find a suitable algorithm
    cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
        weightLayout, inputLayout, outputLayout, outputLayout,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d, gelu: %d\n",
            B, T, C, OC, has_bias, has_gelu);
        exit(EXIT_FAILURE);
    }

    // call the matmul
    const float alpha = 1.0f, beta = 0.0f;
    cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
        &alpha, weight, weightLayout, inp, inputLayout, &beta,
        out, outputLayout, out, outputLayout, &heuristic.algo,
        cublaslt_workspace, cublaslt_workspace_size, 0));

    // cleanups
    cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
    cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
    cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}

// kernel version dispatch
void matmul_forward(int kernel_num,
                    float* out,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C, int OC,
                    const int sqrt_block_size) {
    switch (kernel_num) {
        case 1:
            matmul_forward1(out, inp, weight, bias, B, T, C, OC, sqrt_block_size);
            break;
        case 2:
            matmul_forward2(out, inp, weight, bias, B, T, C, OC, sqrt_block_size);
            break;
        case 3:
            matmul_forward3(out, inp, weight, bias, B, T, C, OC);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int OC = 768 * 4; // expansion of 4, e.g. in the MLP

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * OC * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(OC * C);
    float* bias = make_random_float(OC);

    // move to GPU
    float* d_out;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, OC * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * OC * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, OC * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    matmul_forward_cpu(out, inp, weight, bias, B, T, C, OC);

    // time the kernel at different block sizes
    int sqrt_block_sizes[] = {4, 8, 16, 32};

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++) {
        int sqrt_block_size = sqrt_block_sizes[j];
        printf("Checking block size %d x %d.\n", sqrt_block_size, sqrt_block_size);
        matmul_forward(kernel_num, d_out, d_inp, d_weight, d_bias, B, T, C, OC, sqrt_block_size);
        validate_result(d_out, out, "out", B * T * OC, 1e-1f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++) {
        int sqrt_block_size = sqrt_block_sizes[j];

        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, matmul_forward,
                                              kernel_num, d_out, d_inp, d_weight, d_bias,
                                              B, T, C, OC, sqrt_block_size);

        // napkin math: estimate the flops achieved
        // e.g. A100 40GB PCIe is advertised at 19.5 TFLOPS fp32
        float tflops = (float)B * T * C * OC * 2 / elapsed_time * 1e3f / 1e12f;
        printf("sqrt_block_size %4d | time %.4f ms | tflops %.2f\n", sqrt_block_size, elapsed_time, tflops);
    }

    // free memory
    free(out);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    return 0;
}