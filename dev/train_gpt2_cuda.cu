
#include <hip/hip_runtime.h>
extern "C" __global__  void matmul_forward_kernel(float* out,
    const float* inp, const float* weight, const float* bias,
    int BT, int C, int OC) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // in the naive kernel, every thread handles one element of out
    int bt = blockIdx.x * blockDim.x + threadIdx.x;
    int oc = blockIdx.y * blockDim.y + threadIdx.y;
    if (bt < BT && oc < OC) {
        float val = (bias != NULL) ? bias[oc] : 0.0f;
        const float* wrow = weight + oc * C;
        const float* inp_bt = inp + bt * C;
        for (int i = 0; i < C; i++) {
            val += inp_bt[i] * wrow[i];
        }
        out[bt * OC + oc] = val;
    }
}

extern "C" __global__  void layernorm_forward_kernel(float* out, float* mean, float* rstd,
    const float* inp, const float* weight, const float* bias,
    int N, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float eps = 1e-5f;

    if (idx < N) {
        // seek to the input position inp[idx,:]
        const float* x = inp + idx * C;
        // calculate the mean
        float m = 0.0f;
        for (int i = 0; i < C; i++) {
            m += x[i];
        }
        m = m / C;
        // calculate the variance (without any bias correction)
        float v = 0.0f;
        for (int i = 0; i < C; i++) {
            float xshift = x[i] - m;
            v += xshift * xshift;
        }
        v = v / C;
        // calculate the rstd
        float s = 1.0f / sqrtf(v + eps);
        // seek to the output position in out[idx,:]
        float* out_idx = out + idx * C;
        for (int i = 0; i < C; i++) {
            float n = (s * (x[i] - m)); // normalized output
            float o = n * weight[i] + bias[i]; // scale and shift it
            out_idx[i] = o; // write
        }
        // cache the mean and rstd for the backward pass later
        mean[idx] = m;
        rstd[idx] = s;
    }
}

extern "C" __global__  void residual_forward_kernel(float* out, const float* inp1, const float* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = __ldcs(&inp1[idx]) + __ldcs(&inp2[idx]);
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / 3.14159265358979323846)
extern "C" __global__  void gelu_kernel(float* out, const float* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}