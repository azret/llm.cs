
#include <hip/hip_runtime.h>
#ifndef _HUGE_ENUF
#define _HUGE_ENUF  1e+300  // _HUGE_ENUF*_HUGE_ENUF must overflow
#endif

#ifndef INFINITY
#define INFINITY   ((float)(_HUGE_ENUF * _HUGE_ENUF))
#endif

extern "C" __global__ void attention_query_key_kernel(float* preatt, const float* inp,
    int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) {
            // autoregressive mask
            preatt[idx] = -INFINITY;
            return;
        }
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);

        int C3 = C * 3;
        int hs = C / NH; // head size
        const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
        const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = 0; i < hs; i++) {
            val += query_t[i] * key_t2[i];
        }
        val *= 1.0 / sqrtf(hs);

        preatt[idx] = val;
    }
}

extern "C" __global__ void attention_softmax_kernel(float* att, const float* preatt,
    int B, int T, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        const float* preatt_bth = preatt + b * NH * T * T + h * T * T + t * T;
        float* att_bth = att + b * NH * T * T + h * T * T + t * T;

        // find maxval
        float maxval = -10000.0f; // TODO something better
        for (int t2 = 0; t2 <= t; t2++) {
            if (preatt_bth[t2] > maxval) {
                maxval = preatt_bth[t2];
            }
        }

        // calculate the exp and keep track of sum
        float expsum = 0.0f;
        for (int t2 = 0; t2 <= t; t2++) {
            float expv = expf(preatt_bth[t2] - maxval);
            expsum += expv;
            att_bth[t2] = expv;
        }
        float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

        // normalize to get the softmax
        for (int t2 = 0; t2 < T; t2++) {
            if (t2 <= t) {
                att_bth[t2] *= expsum_inv;
            }
            else {
                // causal attention mask. not strictly necessary to set to zero here
                // only doing this explicitly for debugging and checking to PyTorch
                att_bth[t2] = 0.0f;
            }
        }
    }
}

extern "C" __global__ void attention_value_kernel(float* out, const float* att, const float* inp,
    int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        int C3 = C * 3;
        int hs = C / NH; // head size

        float* out_bth = out + b * T * C + t * C + h * hs;
        const float* att_bth = att + b * NH * T * T + h * T * T + t * T;

        for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
        for (int t2 = 0; t2 <= t; t2++) {
            const  float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C * 2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            for (int i = 0; i < hs; i++) {
                out_bth[i] += att_btht2 * value_t2[i];
            }
        }
    }
}

extern "C" __global__ void encoder_forward_kernel(float* out,
    const int* inp, const float* wte, const float* wpe,
    int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        float* out_btc = out + b * T * C + t * C + c;
        const float* wte_ix = wte + ix * C + c;
        const float* wpe_tc = wpe + t * C + c;
        *out_btc = *wte_ix + *wpe_tc;
    }
}

extern "C" __global__  void matmul_forward_kernel(float* out,
    const float* inp, const float* weight, const float* bias,
    int BT, int C, int OC) {
    int bt = blockIdx.x * blockDim.x + threadIdx.x;
    int oc = blockIdx.y * blockDim.y + threadIdx.y;
    if (bt < BT && oc < OC) {
        float val = (bias != NULL) ? bias[oc] : 0.0f;
        const float* wrow = weight + oc * C;
        const float* inp_bt = inp + bt * C;
        for (int i = 0; i < C; i++) {
            val += inp_bt[i] * wrow[i];
        }
        out[bt * OC + oc] = val;
    }
}

extern "C" __global__  void layernorm_forward_kernel(float* out, float* mean, float* rstd,
    const float* inp, const float* weight, const float* bias,
    int N, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float eps = 1e-5f;

    if (idx < N) {
        // seek to the input position inp[idx,:]
        const float* x = inp + idx * C;
        // calculate the mean
        float m = 0.0f;
        for (int i = 0; i < C; i++) {
            m += x[i];
        }
        m = m / C;
        // calculate the variance (without any bias correction)
        float v = 0.0f;
        for (int i = 0; i < C; i++) {
            float xshift = x[i] - m;
            v += xshift * xshift;
        }
        v = v / C;
        // calculate the rstd
        float s = 1.0f / sqrtf(v + eps);
        // seek to the output position in out[idx,:]
        float* out_idx = out + idx * C;
        for (int i = 0; i < C; i++) {
            float n = (s * (x[i] - m)); // normalized output
            float o = n * weight[i] + bias[i]; // scale and shift it
            out_idx[i] = o; // write
        }
        // cache the mean and rstd for the backward pass later
        mean[idx] = m;
        rstd[idx] = s;
    }
}

extern "C" __global__  void residual_forward_kernel(float* out, const float* inp1, const float* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = __ldcs(&inp1[idx]) + __ldcs(&inp2[idx]);
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / 3.14159265358979323846)
extern "C" __global__  void gelu_kernel(float* out, const float* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}